
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x]; 
}


void random_ints(int *nums, int size){
	
	for (int i = 0; i < size; i++){
		nums[i] = 1;
	}
}


#define N 512
int main(void){
	
	int *a, *b, *c;			//host copies of a, b, c
	int *d_a, *d_b, *d_c;	//device copies of a, b, c
	int size = N * sizeof(int);

	//Allocate space for device copies of a, b, c
	hipMalloc((void **) &d_a, size);	//take in address of pointer 
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);


	// Alloc space for host copies of a, b, c and setup input values 
	a = (int *)malloc(size);
	random_ints(a, N);

	b = (int *)malloc(size);
	random_ints(b, N); 

	c = (int *)malloc(size);

	//copy inputs to device 
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	//launch add() kernel on GPU with N blocks
	add<<<N,1>>>(d_a, d_b, d_c);

	//copy result back to host 
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	//print results
	for (int i = 0; i < N; i++){
		printf("c[%d]=%d\n", i, c[i]);
	}

	
	//Cleanup 
	free(a);
	free(b);
	free(c);

	hipFree(d_a); 
	hipFree(d_b);
	hipFree(d_c);
	return 0;

}