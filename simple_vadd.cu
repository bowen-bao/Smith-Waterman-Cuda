#include "hip/hip_runtime.h"
__global__ void add(int *a, int *b, int *c){
	*c = *a + *b;
}

int main(void){
	
	int a, b, c;			//host copies of a, b, c
	int *d_a, *d_b, *d_c	//device copies of a, b, c
	int size = sizeof(int)

	//Allocate space for device copies of a, b, c
	hipMalloc((void **) &d_a, size);	//take in address of pointer 
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	//set up input values 
	a = 2;
	b = 7;

	//copy inputs to device 
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &a, size, hipMemcpyHostToDevice);

	//launch add() kernel on GPU
	add<<<1,1>>>(d_a, d_b, d_c)

	//copy result back to host 
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	//Cleanup 
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;

}